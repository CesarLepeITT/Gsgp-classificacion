#include "hip/hip_runtime.h"
/*<one line to give the program's name and a brief idea of what it does.>
    Copyright (C) 2020 José Manuel Muñoz Contreras, Leonardo Trujillo, Daniel E. Hernandez, Perla Juárez Smith

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

     This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU Affero General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

//! \file   GsgpCuda.cu
//! \brief  file containing the main with the geometric semantic genetic programming algorithm
//! \author Jose Manuel Muñoz Contreras, Leonardo Trujillo, Daniel E. Hernandez, Perla Juárez Smith
//! \date   created on 25/01/2020
#include "GsgpCuda.cpp"

/*!
* \fn       int main(int argc, const char **argv)
* \brief    main method that runs the GSGP algorithm and test the best model generate by GSGP-CUDA
* \param    int argc: number of parameters of the program
* \param    const char **argv: array of strings that contains the parameters of the program
* \return   int: 0 if the program ends without errors
* \date     25/01/2020
* \author   Jose Manuel Muñoz Contreras, Leonardo Trujillo, Daniel E. Hernandez, Perla Juárez Smith
* \file     GsgpCuda.cu
*/
int main(int argc, char **argv){
    hipSetDevice(0); /*!< Select a GPU device*/    
    srand(time(NULL)); /*!< Initialization of the seed for the generation of random numbers*/

    char trainFile[500]="";    /*!< Name of the train file*/
    char testFile[500]="";     /*!< Name of the test file*/
    char output_model[500]=""; /*!< Name of output files*/
    char pathTrace[500]="";    /*!< Name of the file trace of best model*/
    char path_test[500]="";    /*!< Name of the file with unsseen test instances*/
    char pathOutFile[500]="";  /*!< Name of the file to output values*/
    char pathIniFile[500]="";  /*!< Name of the file to output values*/
    for (int i=1; i<argc-1; i++){
        if(strncmp(argv[i],"-train_file",10) == 0) {
            strcat(trainFile,argv[++i]);
        }else if (strncmp(argv[i],"-test_file",10) == 0) {
            strcat(testFile,argv[++i]);
        }else if (strncmp(argv[i],"-output_model",10)==0) {
            strcat(output_model,argv[++i]);
        }else if (strncmp(argv[i],"-model",10)==0) {
            strcat(pathTrace,argv[++i]);
        }else if (strncmp(argv[i],"-input_data",10)==0) {
            strcat(path_test,argv[++i]);
        }else if (strncmp(argv[i],"-prediction_output",10)==0) {
            strcat(pathOutFile,argv[++i]);
        }else if (strncmp(argv[i],"-log_path",10)==0) {
            strcat(pathIniFile,argv[++i]);
        }     
    }
    
    std::string trainFile_s(trainFile);
 
    std::string testFile_s(testFile);
 
    std::string outputNameFiles(output_model); ///*!< Name of file for save the output files*/

    std::string pathIni(pathIniFile);

    std::string la (pathTrace);

    readConfigFile(pathIni,&config); /*!< reading the parameters of the algorithm */

    std::string logPath (config.logPath); /* Path of directory for data files and log files generated in execution */
        
    std::string namePopulation = "_initialPopulation.csv"; /*!< Name of file for save the initial population  */

    namePopulation = outputNameFiles + namePopulation;

    std::string nameRandomTrees = "_randomTrees.csv"; /*!< name of file for save the random trees */
    
    nameRandomTrees = outputNameFiles + nameRandomTrees; 

    if (!trainFile_s.empty() && testFile_s.empty()){

        printf("\n Evolution with GsgpCuda \n\n");
        
        countInputFile(trainFile, nrow, nvar); ///Counting the number of rows and variables of the train file
        
        nvar--; 
        printf("%i %i \n", nrow,nvar);
        individualLength = config.maxIndividualLength; /*!< Variable that stores maximum depth for individuals */

        sizeMemIndividuals = sizeof(float) * config.populationSize; /*!< Variable that stores size in bytes of the number of individuals in the initial population*/

        sizeMemNormalize = sizeof(float) * config.populationSize*4;

        twoSizeMemPopulation = sizeof(float) * (config.populationSize*2); /*!< Variable that stores twice the size in bytes of an initial population to store random numbers*/

        sizeMemPopulation = sizeof(float) * config.populationSize * individualLength; /*!< Variable that stores size in bytes for initial population*/
        
        twoSizePopulation = (config.populationSize*2); /*!< Variable storing twice the initial population of individuals to generate random positions*/

        sizeMemSemanticTrain = sizeof(float)*(config.populationSize*nrow); /*!< Variable that stores the size in bytes of semantics for the entire population with training data*/

        sizeMemDataTrain = sizeof(float)*(nrow*nvar); /*!< Variable that stores the size in bytes the size of the training data*/

        sizeElementsSemanticTrain = (config.populationSize*nrow); /*!< Variable that stores training data elements*/

        vectorTracesMem = (sizeof(entry_)*config.numberGenerations*config.populationSize); /*!< Variable that stores the size in bytes of the structure to store the survival record*/

        std::string logPath (config.logPath); /* Path of directory for data files and log files generated in execution */

        std::string namePopulation = "_initialPopulation.csv"; /*!< Name of file for save the initial population  */

        namePopulation = outputNameFiles + namePopulation;

        std::string nameRandomTrees = "_randomTrees.csv"; /*!< name of file for save the random trees */
        
        nameRandomTrees = outputNameFiles + nameRandomTrees;

        /* Check if log and data diectories exists */
        checkDirectoryPath(logPath);
        
        float executionTime = 0, initialitionTimePopulation = 0, timeComputeSemantics = 0, generationTime = 0; /*!< Variables that store the time in milliseconds between the events mark1 and mark2.*/
        std::string timeExecution1 = "_processing_time"; /*!< Variable name structure responsible for indicating the run*/
        std::string timeExecution2 = ".csv"; /*!< Variable name structure responsible for indicating the file extension*/
        timeExecution1 = logPath + outputNameFiles + timeExecution1 + timeExecution2; /*!< Variable that stores file name matching*/
        std::ofstream times(timeExecution1,ios::out); /*!< pointer to the timeExecution1 file that contains the time consumed by the different algorithm modules*/
 
        hipEvent_t startRun, stopRun;  /*!< Variable used to create a start mark and a stop mark to create events*/
        hipEventCreate(&startRun);     /*!< function that initializes the start event*/
        hipEventCreate(&stopRun);      /*!< function that initializes the stop event*/

        hiprandState_t* states; /*!< CUDA's random number library uses hiprandState_t to keep track of the seed value we will store a random state for every thread*/
        hipMalloc((void**) &states, config.populationSize * sizeof(hiprandState_t)); /*!< allocate space on the GPU for the random states*/
        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, init, 0, config.populationSize); /*!< heuristic function used to choose a good block size is to aim at high occupancy*/
        printf("%i %i %i \n", minGridSize, blockSize, gridSize);
        gridSize = (config.populationSize + blockSize - 1) / blockSize; /*!< round up according to array size*/
        init<<<gridSize, blockSize>>>(time(0), states); /*!< invoke the GPU to initialize all of the random states*/

        hipEventRecord(startRun);     
        std::string fitnessTrain  = "_fitnestrain"; /**/
        std::string fitnessTrain2 = ".csv"; /**/
        fitnessTrain = logPath + outputNameFiles + fitnessTrain + fitnessTrain2; /**/
        std::ofstream fitTraining(fitnessTrain,ios::out); /*!< pointer to the file fitnesstrain.csv containing the training fitness of the best individual at each generation*/

        hipblasHandle_t handle; /*!< the handle to the cuBLAS library context*/
        hipblasCreate(&handle); /*!< initialized using the function and is explicitly passed to every subsequent library function call*/
        hNormalizeData = (float *)malloc(sizeMemNormalize); 
        hInitialPopulation = (float *)malloc(sizeMemPopulation); 
        hRandomTrees = (float *)malloc(sizeMemPopulation); 
        hipMalloc((void **)&dRandomTrees, sizeMemPopulation); 
        hipMalloc((void **)&dInitialPopulation, sizeMemPopulation);
        hipMallocManaged(&vectorTraces,vectorTracesMem);
        hipMallocManaged(&uDataTrain, sizeMemDataTrain);     
        hipMallocManaged(&uDataTrainTarget, sizeof(float)*nrow); 
        hipMallocManaged(&uFit, sizeMemIndividuals);  
        hipMallocManaged(&uSemanticTrainCases,sizeMemSemanticTrain);       
        hipMallocManaged(&uSemanticRandomTrees,sizeMemSemanticTrain);      
        hipMalloc((void**)&uPushGenes, sizeMemIndividuals);
        hipMalloc((void**)&uStackInd, sizeMemPopulation); 
        hipMalloc((void**)&dNormalizeData, sizeMemNormalize); 
        if(config.oms==1){
            hipMallocManaged(&uDifferenceRandomTrees,sizeMemSemanticTrain);
            hipMallocManaged(&uDifferenceRtPow,sizeMemSemanticTrain);
            hipMallocManaged(&inverse,sizeMemSemanticTrain);
            hipMallocManaged(&oms, sizeMemIndividuals);
        }                     
       
        readInpuDataTrain(trainFile, uDataTrain, uDataTrainTarget, nrow, nvar); /// load set data train **/ 
        gridSize = (config.populationSize + blockSize - 1) / blockSize; /*!< round up according to array size*/            
        
        hipEvent_t startInitialPop, stopInitialPop; /*!< this section declares and initializes the Variables for the events and captures the time elapsed in the initialization of the initial population in the GPU*/
        hipEventCreate(&startInitialPop);
        hipEventCreate(&stopInitialPop);
        hipEventRecord(startInitialPop);

        ///invokes the GPU to initialize the initial population
        initializePopulation<<< gridSize, blockSize >>>(dInitialPopulation, nvar, individualLength, states, config.maxRandomConstant, 4, config.functionRatio, config.variableRatio);
        cudaErrorCheck("initializePopulation");
        
        hipEventRecord(stopInitialPop);
        hipEventSynchronize(stopInitialPop);
        hipEventElapsedTime(&initialitionTimePopulation, startInitialPop, stopInitialPop);
        hipEventDestroy(startInitialPop);
        hipEventDestroy(stopInitialPop);    
        ///*!<return the initial population of the device to the host*/
        hipMemcpy(hInitialPopulation, dInitialPopulation, sizeMemPopulation, hipMemcpyDeviceToHost);    
        saveIndividuals(logPath,hInitialPopulation, namePopulation, individualLength,config.populationSize);  

        ///*!< invokes the GPU to initialize the random trees*/
        initializePopulation<<< gridSize, blockSize >>>(dRandomTrees, nvar, individualLength, states, config.maxRandomConstant,4,config.functionRatio, config.variableRatio);    
        cudaErrorCheck("initializePopulation");    
        ///*!<return the initial population of the device to the host*/
        hipMemcpy(hRandomTrees, dRandomTrees,sizeMemPopulation, hipMemcpyDeviceToHost);
        saveIndividuals(logPath,hRandomTrees, nameRandomTrees,individualLength,config.populationSize); 
        hipEvent_t startComputeSemantics, stopComputeSemantics; /*!< This section declares and initializes the Variables for the events and captures the time elapsed in the interpretation of the initial population in the GPU*/
        hipEventCreate(&startComputeSemantics);
        hipEventCreate(&stopComputeSemantics);
        hipEventRecord(startComputeSemantics);    
        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, computeSemantics, 0, config.populationSize); /*!< heuristic function used to choose a good block size is to aim at high occupancy*/
        gridSize = (config.populationSize + blockSize - 1) / blockSize; /*!< round up according to array size*/      
     
        /*!< invokes the GPU to interpret the initial population with data train*/
        computeSemantics<<< gridSize, blockSize >>>(dInitialPopulation, uSemanticTrainCases, individualLength, uDataTrain, nrow, nvar, uPushGenes, uStackInd,config.protected_division);
        cudaErrorCheck("computeSemantics");            
        /*!< invokes the GPU to interpret the random trees with data train*/
        computeSemantics<<< gridSize, blockSize >>>(dRandomTrees, uSemanticRandomTrees, individualLength, uDataTrain, nrow, nvar, uPushGenes, uStackInd,config.protected_division);
        // hipDeviceSynchronize();
        // saveSemantics(logPath,uSemanticRandomTrees,config.populationSize,outputNameFiles,nrow,0);
        cudaErrorCheck("computeSemantics");    
        if(config.normalize==1){
            
            /*!< invokes the GPU to interpret the random trees with data train*/
            computeNormalizeData<<< gridSize, blockSize >>>( uSemanticRandomTrees, dNormalizeData, nrow);
            cudaErrorCheck("computeNormalizeData");

            computeNormalizeSemantic<<< gridSize, blockSize >>>(uSemanticRandomTrees, dNormalizeData, nrow, config.do_min_max,1);
            cudaErrorCheck("computeNormalizeSemantic");

            hipMemcpy(hNormalizeData, dNormalizeData,sizeMemNormalize,hipMemcpyDeviceToHost);
            saveDataNormalize(logPath,hNormalizeData,config.populationSize,outputNameFiles);
        }
        hipDeviceSynchronize();
        saveSemantics(logPath,uSemanticRandomTrees,config.populationSize,outputNameFiles,nrow,1);
        hipEventRecord(stopComputeSemantics);
        hipEventSynchronize(stopComputeSemantics);
        hipEventElapsedTime(&timeComputeSemantics, startComputeSemantics, stopComputeSemantics);
        hipEventDestroy(startComputeSemantics);
        hipEventDestroy(stopComputeSemantics);  

        /*!< memory is deallocated for training data and auxiliary vectors for the interpreter*/
        hipFree(uDataTrain);
        // hipFree(uStackInd);
        // hipFree(uPushGenes);       
    
        if(config.errorFunction==0){
            hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, computeError, 0, config.populationSize); 
            gridSize = (config.populationSize + blockSize - 1) / blockSize;  
            if((gridSize*blockSize)>config.populationSize){
                blockSize = 32;
                gridSize = (config.populationSize + blockSize - 1) / blockSize;
            }
            if((gridSize*blockSize)>config.populationSize){
                blockSize = 16;
                gridSize = (config.populationSize + blockSize - 1) / blockSize;
            } 
            /*!< invokes the GPU to calculate the error (RMSE) the initial population*/
            computeError<<< gridSize, blockSize>>>(uSemanticTrainCases, uDataTrainTarget, uFit, nrow);
            cudaErrorCheck("computeError");
            
            hipblasIsamin(handle, config.populationSize, uFit, incx1, &result);
            indexBestIndividual = result-1;
        
        }else if(config.errorFunction==1){   
            hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, computeRo, 0, config.populationSize); 
            gridSize = (config.populationSize + blockSize - 1) / blockSize;   
            targetMean = findMean(uDataTrainTarget,nrow);
            if((gridSize*blockSize)>config.populationSize){
                blockSize = 32;
                gridSize = (config.populationSize + blockSize - 1) / blockSize;
            }else if((gridSize*blockSize)>config.populationSize){
                blockSize = 16;
                gridSize = (config.populationSize + blockSize - 1) / blockSize;
            }            
            computeRo<<<gridSize, blockSize>>>(uSemanticTrainCases,uDataTrainTarget, uFit,nrow, targetMean);
            cudaErrorCheck("computeRo");
            hipblasIsamax(handle, config.populationSize, uFit, incx1, &result);
            indexBestIndividual = result-1;
        }

        /*!< function is necessary so that the CPU does not continue with the execution of the program and allows to capture the fitness*/
        hipDeviceSynchronize();
        
        /*!< writing the  training fitness of the best individual on the file fitnesstrain.csv*/
        if(config.errorFunction==0){
            fitTraining << 0 << "," <<uFit[indexBestIndividual]<<endl;    
        }else if (config.errorFunction==1){
            fitR2 = uFit[indexBestIndividual] - (-4000*(-1));
            fitTraining << 0 << "," << fitR2 <<endl;
        }

        hipMallocManaged(&uSemanticTrainCasesNew,sizeMemSemanticTrain);
        hipMallocManaged(&uFitNew, sizeMemPopulation);
        hipEvent_t startGsgp, stopGsgp;
        hipEventCreate(&startGsgp);
        hipEventCreate(&stopGsgp);          
        
        hiprandState_t* State;
        hipMalloc((void**) &State, (twoSizePopulation) * sizeof(hiprandState_t));
        hipMallocManaged(&indexRandomTrees,twoSizeMemPopulation);
        hipMallocManaged(&mutationStep,sizeMemPopulation); 

        hiprandState_t* statesMutationStep;
        hipMalloc((void**) &statesMutationStep, (sizeMemPopulation) * sizeof(hiprandState_t));
        int index =0;   
        /*!< main GSGP cycle*/
        for ( int generation=1; generation<=config.numberGenerations; generation++){

            /*!< register execution time*/
            hipEventRecord(startGsgp);
            gridSize =0, blockSize=0;
            index = generation-1;
            
            hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, init, 0, twoSizePopulation);
            gridSize = (twoSizePopulation + blockSize - 1) / blockSize;
            
            init<<<gridSize, blockSize>>>(time(NULL)*index, State); /*!< initializes the random number generator*/
            cudaErrorCheck("init");
            
            /*!< invokes the GPU to initialize the random positions of the random trees*/
            hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, initializeIndexRandomTrees, 0, twoSizePopulation);
            gridSize = (twoSizePopulation + blockSize - 1) / blockSize;

            initializeIndexRandomTrees<<<gridSize,blockSize >>>( config.populationSize, indexRandomTrees, State );
            cudaErrorCheck("initializeIndexRandomTrees");
                                  
            if(config.oms==0){

                hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, init, 0, config.populationSize);
                gridSize = (config.populationSize + blockSize - 1) / blockSize;
                
                init<<<gridSize, blockSize>>>(time(NULL)*index, statesMutationStep); /*!< initializes the random number generator*/
                cudaErrorCheck("init");

                /*!< invokes the GPU to initialize the random positions of the random trees*/
                hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, initializeMutationStep, 0, config.populationSize);
                gridSize = (config.populationSize + blockSize - 1) / blockSize;

                initializeMutationStep<<<gridSize,blockSize >>>(mutationStep, statesMutationStep);
                cudaErrorCheck("initializeMutationStep");
                
                hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, geometricSemanticMutation, 0, sizeElementsSemanticTrain);
                gridSize = (sizeElementsSemanticTrain + blockSize - 1) / blockSize;
                
                if((gridSize*blockSize)>sizeElementsSemanticTrain){
                    blockSize = 32;
                    gridSize = (sizeElementsSemanticTrain + blockSize - 1) / blockSize;
                }else if((gridSize*blockSize)>sizeElementsSemanticTrain){
                    blockSize = 16;
                    gridSize = (sizeElementsSemanticTrain + blockSize - 1) / blockSize;
                }
                
                /*!< geometric semantic mutation with semantic train*/
                geometricSemanticMutation<<< gridSize, blockSize >>>(uSemanticTrainCases, uSemanticRandomTrees,uSemanticTrainCasesNew,
                config.populationSize, nrow, sizeElementsSemanticTrain, generation, indexRandomTrees, vectorTraces, index, mutationStep, config.sigmoid,config.normalize);
                cudaErrorCheck("geometricSemanticMutation");
                            
            }else if(config.oms==1){
                // printf("Dentro de la opccion de OMS \n");
                
                hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, init, 0, config.populationSize);
                gridSize = (config.populationSize + blockSize - 1) / blockSize;
                
                init<<<gridSize, blockSize>>>(time(NULL)*index, statesMutationStep); /*!< initializes the random number generator*/
                cudaErrorCheck("init");

                hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, initializeMutationStep, 0, config.populationSize);
                gridSize = (config.populationSize + blockSize - 1) / blockSize;

                initializeMutationStep<<<gridSize,blockSize >>>(mutationStep, statesMutationStep);
                cudaErrorCheck("initializeMutationStep");

                hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, diferenceRt, 0, sizeElementsSemanticTrain);
                gridSize = (sizeElementsSemanticTrain + blockSize - 1) / blockSize;
                if((gridSize*blockSize)>sizeElementsSemanticTrain){
                    blockSize = 32;
                    gridSize = (sizeElementsSemanticTrain + blockSize - 1) / blockSize;                    
                }else if((gridSize*blockSize)>sizeElementsSemanticTrain){
                    blockSize = 16;
                    gridSize = (sizeElementsSemanticTrain + blockSize - 1) / blockSize;
                }
                
                diferenceRt<<<gridSize,blockSize>>>(nrow,config.populationSize,uSemanticRandomTrees, indexRandomTrees,uDifferenceRandomTrees,config.sigmoid,config.normalize);
                
                rt2<<<gridSize,blockSize>>>(uDifferenceRandomTrees, uDifferenceRtPow);
                
                inverseVector<<<gridSize,blockSize>>>(uDifferenceRandomTrees, uDifferenceRtPow,nrow,inverse);

                hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, makeOms, 0, config.populationSize); 
                gridSize = (config.populationSize + blockSize - 1) / blockSize; 
                
                makeOms<<<gridSize,blockSize>>>(uSemanticTrainCases,inverse,uDataTrainTarget,oms,nrow,config.populationSize, mutationStep);
                
                hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, geometricSemanticMutation, 0, sizeElementsSemanticTrain);
                gridSize = (sizeElementsSemanticTrain + blockSize - 1) / blockSize;
                
                if((gridSize*blockSize)>sizeElementsSemanticTrain){
                    blockSize = 32;
                    gridSize = (sizeElementsSemanticTrain + blockSize - 1) / blockSize;
                 
                }else if((gridSize*blockSize)>sizeElementsSemanticTrain){
                    blockSize = 16;
                    gridSize = (sizeElementsSemanticTrain + blockSize - 1) / blockSize;
                }

                geometricSemanticMutation<<< gridSize, blockSize >>>(uSemanticTrainCases, uSemanticRandomTrees,uSemanticTrainCasesNew,
                config.populationSize, nrow, sizeElementsSemanticTrain, generation, indexRandomTrees, vectorTraces, index, oms, config.sigmoid,config.normalize);
                cudaErrorCheck("geometricSemanticMutation");
            }

            if(config.errorFunction==0){
        
                hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, computeError, 0, config.populationSize); 
                gridSize = (config.populationSize + blockSize - 1) / blockSize;
                
                if((gridSize*blockSize)>config.populationSize){
                    blockSize = 32;
                    gridSize = (config.populationSize + blockSize - 1) / blockSize;
                }else if((gridSize*blockSize)>config.populationSize){
                    blockSize = 16;
                    gridSize = (config.populationSize + blockSize - 1) / blockSize;
                } 
                
                /*!< invokes the GPU to calculate the error (RMSE) the initial population*/
                computeError<<< gridSize, blockSize>>>(uSemanticTrainCasesNew, uDataTrainTarget, uFitNew, nrow);
                cudaErrorCheck("computeError");
                hipblasIsamin(handle, config.populationSize, uFitNew, incx1, &result);
                indexBestOffspring = result-1;
                hipblasIsamax(handle, config.populationSize, uFitNew, incxWorst, &resultWorst);
                indexWorstOffspring = resultWorst-1;
                
            }else if(config.errorFunction==1){    
                
                hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, computeError, 0, config.populationSize); 
                gridSize = (config.populationSize + blockSize - 1) / blockSize;
                targetMean = findMean(uDataTrainTarget,nrow);
                if((gridSize*blockSize)>config.populationSize){
                    blockSize = 32;
                    gridSize = (config.populationSize + blockSize - 1) / blockSize;
                }else if((gridSize*blockSize)>config.populationSize){
                    blockSize = 16;
                    gridSize = (config.populationSize + blockSize - 1) / blockSize;
                }            
                computeRo<<<gridSize, blockSize>>>(uSemanticTrainCasesNew,uDataTrainTarget, uFitNew,nrow, targetMean);
                hipblasIsamax(handle, config.populationSize, uFitNew, incxWorst, &resultWorst);
                indexBestOffspring = resultWorst-1;
                hipblasIsamin(handle, config.populationSize, uFitNew, incxBestOffspring, &resultBestOffspring);
                indexWorstOffspring = resultBestOffspring-1;
            }

            /*!< set byte values*/
            hipMemset(indexRandomTrees,0,twoSizeMemPopulation);
            hipMemset(mutationStep,0,sizeMemPopulation);
            hipDeviceSynchronize();
            
            /*!< this section performs survival by updating the semantic and fitness vectors respectively*/
            if(config.errorFunction==0){
                if(uFitNew[indexBestOffspring] > uFit[indexBestIndividual]){
                    // printf("Pasa el padre RMSE %i valor del mejor hijo %f vs del mejor padre %f\n", indexBestIndividual, uFitNew[indexBestOffspring],uFit[indexBestIndividual]);
                    vectorTraces[(index*config.populationSize)+indexWorstOffspring].firstParent = indexBestIndividual;
                    vectorTraces[(index*config.populationSize)+indexWorstOffspring].secondParent = indexBestIndividual;
                    vectorTraces[(index*config.populationSize)+indexWorstOffspring].number = indexBestIndividual;
                    vectorTraces[(index*config.populationSize)+indexWorstOffspring].event = -1;
                    vectorTraces[(index*config.populationSize)+indexWorstOffspring].newIndividual = indexBestIndividual;
                    vectorTraces[(index*config.populationSize)+indexWorstOffspring].mark= 0;
                    vectorTraces[(index*config.populationSize)+indexWorstOffspring].mutStep = 0;
    
                    for (int i = 0; i < nrow; ++i){
                        uSemanticTrainCasesNew[indexWorstOffspring*nrow+i] = uSemanticTrainCases[indexBestIndividual*nrow+i];
                    }
    
                    uFitNew[indexWorstOffspring] = uFit[indexBestIndividual];
                    tempFitnes = uFit;
                    uFit = uFitNew;
                    uFitNew = tempFitnes;
                    tempSemantic = uSemanticTrainCases;
                    uSemanticTrainCases = uSemanticTrainCasesNew;
                    uSemanticTrainCasesNew = tempSemantic;
 
                    indexBestIndividual = indexWorstOffspring;
                }else{
                    // printf("Pasa el hijo RMSE %i \n", indexBestOffspring);
                    tempFitnes = uFit;
                    uFit = uFitNew;
                    uFitNew = tempFitnes;
                    tempSemantic = uSemanticTrainCases;
                    uSemanticTrainCases = uSemanticTrainCasesNew;
                    uSemanticTrainCasesNew = tempSemantic;
                    indexBestIndividual = indexBestOffspring;
                }
            } else if(config.errorFunction==1){
                if(uFitNew[indexBestOffspring] < uFit[indexBestIndividual]){
                    // printf("Pasa el padre %i \n", indexBestIndividual);
                    vectorTraces[(index*config.populationSize)+indexWorstOffspring].firstParent = indexBestIndividual;
                    vectorTraces[(index*config.populationSize)+indexWorstOffspring].secondParent = indexBestIndividual;
                    vectorTraces[(index*config.populationSize)+indexWorstOffspring].number = indexBestIndividual;
                    vectorTraces[(index*config.populationSize)+indexWorstOffspring].event = -1;
                    vectorTraces[(index*config.populationSize)+indexWorstOffspring].newIndividual = indexBestIndividual;
                    vectorTraces[(index*config.populationSize)+indexWorstOffspring].mark= 0;
                    vectorTraces[(index*config.populationSize)+indexWorstOffspring].mutStep = 0;
    
                    for (int i = 0; i < nrow; ++i){
                        uSemanticTrainCasesNew[indexWorstOffspring*nrow+i] = uSemanticTrainCases[indexBestIndividual*nrow+i];
                    }
    
                    uFitNew[indexWorstOffspring] = uFit[indexBestIndividual];
                    tempFitnes = uFit;
                    uFit = uFitNew;
                    uFitNew = tempFitnes;
                    tempSemantic = uSemanticTrainCases;
                    uSemanticTrainCases = uSemanticTrainCasesNew;
                    uSemanticTrainCasesNew = tempSemantic;
                    indexBestIndividual = indexWorstOffspring;

                }else{
                    // printf("Pasa el hijo %i \n", indexBestOffspring);
                    tempFitnes = uFit;
                    uFit = uFitNew;
                    uFitNew = tempFitnes;
                    tempSemantic = uSemanticTrainCases;
                    uSemanticTrainCases = uSemanticTrainCasesNew;
                    uSemanticTrainCasesNew = tempSemantic;
                    indexBestIndividual = indexBestOffspring;
                } 
            }
            


            /*!< writing the  training fitness of the best individual on the file fitnesstrain.csv*/
            if(config.errorFunction==0){
                fitTraining << generation << "," <<uFit[indexBestIndividual]<<endl;    
            }if (config.errorFunction==1){
                fitR2=0;
                fitR2 = uFit[indexBestIndividual] - (-4000*(-1));
                fitTraining << generation << "," << fitR2 <<endl;    
            }
            hipEventRecord(stopGsgp);
            hipEventSynchronize(stopGsgp);
            hipEventElapsedTime(&generationTime, startGsgp, stopGsgp);    
        }

        markTracesGeneration(vectorTraces, config.populationSize, config.numberGenerations,  indexBestIndividual);
        saveTrace(outputNameFiles,logPath, vectorTraces, config.numberGenerations, config.populationSize);
        
        float *dOutIndividuals;
        int *pushGenesModel;
        float *uStackIndi,*uStackModel, *hostuStackModel, *uStackModelRt, *hostuStackModelRt;
        hipMalloc((void**)&dOutIndividuals, sizeof(float)*(config.populationSize*nrow));
        hipMalloc((void**)&pushGenesModel, sizeMemIndividuals);
        hipMalloc((void**)&uStackModel, sizeMemPopulation);
        hipMalloc((void**)&uStackModelRt, sizeMemPopulation);
        hipMalloc((void**)&uStackIndi, sizeMemPopulation);
        hostuStackModel = (float *)malloc(sizeMemPopulation); 
        hostuStackModelRt = (float *)malloc(sizeMemPopulation); 
        
        /*!< invokes the GPU to interpret the initial population with data train*/
        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, computeSemanticsExpressions, 0, config.populationSize); /*!< heuristic function used to choose a good block size is to aim at high occupancy*/
        gridSize = (config.populationSize + blockSize - 1) / blockSize; /*!< round up according to array size*/      
        
        computeSemanticsExpressions<<< gridSize, blockSize >>>(dInitialPopulation, dOutIndividuals, individualLength, 1, nvar, uPushGenes, pushGenesModel, uStackInd, uStackModel,config.protected_division);
        cudaErrorCheck("computeSemanticsExpressions");
        hipMemcpy(hostuStackModel, uStackModel, sizeMemPopulation, hipMemcpyDeviceToHost);    
        
        computeSemanticsExpressions<<< gridSize, blockSize >>>(dRandomTrees, dOutIndividuals, individualLength, 1, nvar, uPushGenes, pushGenesModel, uStackInd, uStackModelRt,config.protected_division);
        cudaErrorCheck("computeSemanticsExpressions");
        hipMemcpy(hostuStackModelRt, uStackModelRt, sizeMemPopulation, hipMemcpyDeviceToHost);    
        
        int sizeOfIndividuals[config.populationSize];
        int sizeOfRandomTrees[config.populationSize];
        
        //Reconstruccion de expresion matematica de los individuos
        getModel(config, hostuStackModel, hostuStackModelRt, sizeOfIndividuals, sizeOfRandomTrees);
        printf("Paso el modelo \n");
        //Generacion del modelo a partir a de trace
        expre_model(outputNameFiles, logPath, sizeOfIndividuals, sizeOfRandomTrees, config, hNormalizeData);    
        printf("Paso el modelo total\n");
        //Llamada para almacenar los individuos en expresion matematica
        saveModelIndividuals(logPath, outputNameFiles, 0);
        saveModelIndividuals(logPath, outputNameFiles, 1);
        /*!< at the end of the execution  to deallocate memory*/
        free(hNormalizeData);
        hipFree(dNormalizeData);
        hipFree(hostuStackModel);
        hipFree(hostuStackModelRt);
        hipFree(dOutIndividuals);
        hipFree(pushGenesModel);
        hipFree(uStackModel);
        hipFree(uStackModelRt);
        hipFree(uStackIndi);
        hipFree(uStackInd);
        hipFree(uPushGenes); 
        hipFree(indexRandomTrees);
        hipFree(vectorTraces);
        hipblasDestroy(handle);
        hipFree(dInitialPopulation);
        hipFree(dRandomTrees);
        free(hInitialPopulation);
        free(hRandomTrees);
        hipFree(uDataTrainTarget);
        hipFree(uFit);
        hipFree(uFitNew);
        hipFree(uSemanticTrainCases);
        hipFree(uSemanticRandomTrees);
        hipFree(uSemanticTrainCasesNew);
        hipFree(mutationStep);
        if(config.oms==1){
            hipFree(uDifferenceRandomTrees);
            hipFree(uDifferenceRtPow);
            hipFree(inverse);
            hipFree(oms);
        }
        hipEventRecord(stopRun);
        hipEventSynchronize(stopRun);
        hipEventElapsedTime(&executionTime, startRun, stopRun);

         /*!< writing the time execution for stages the algorithm*/
        times << config.populationSize
        << "," << individualLength 
        << "," << nrow 
        << "," << nvar 
        << "," << executionTime/1000
        << "," << initialitionTimePopulation/1000
        << "," << timeComputeSemantics/1000
        << "," << generationTime/1000
        <<endl;
        hipFree(State);
        hipFree(states);
        hipFree(statesMutationStep);
        /*!< all device allocations are removed*/
        hipDeviceReset();
        return 0;
    }else if (!la.empty()) {

        printf("\n Evaluating the model generated by GsgpCuda \n\n");
        
        countInputFile(path_test, nrowTest, nvar);
        // nvar--; 
        printf("Navr en predit %i \n",nvar);
        namePopulation =  logPath +la + namePopulation;
        nameRandomTrees = logPath +la + nameRandomTrees;
        // printf("ini pop %s and rt %s and log %s \n", namePopulation.c_str(), nameRandomTrees.c_str(), logPath.c_str());
        std::string outFile (pathOutFile);
        outFile = logPath + outFile;
        outFile.c_str();
        individualLength = config.maxIndividualLength; /*!< Variable that stores maximum depth for individuals */
        sizeMemPopulation = sizeof(float) * config.populationSize * individualLength; /*!< Variable that stores size in bytes for initial population*/
        sizeMemIndividuals = sizeof(float) * config.populationSize; /*!< Variable that stores size in bytes of the number of individuals in the initial population*/
        sizeMemNormalize = sizeof(float) * config.populationSize*4;
        float *initPopulation, *randomTress, *dInitialPopulation,*dRandomTrees; /*!< This vector pointers to store the individuals of the initial population and random trees */
        initPopulation = (float*)malloc(sizeMemPopulation); /*!<  Variable that stores the size in bytes the initial population */
        randomTress = (float*)malloc(sizeMemPopulation);  /*!< Variable that stores the size in bytes the initial population */

        hipMalloc((void **)&dRandomTrees, sizeMemPopulation); 
        hipMalloc((void **)&dInitialPopulation, sizeMemPopulation);

        readPopulation(initPopulation, randomTress, config.populationSize, individualLength, logPath, namePopulation, nameRandomTrees);

        ///*!<return the initial population of the device to the host*/
        hipMemcpy(dInitialPopulation, initPopulation, sizeMemPopulation, hipMemcpyHostToDevice); 
        hipMemcpy(dRandomTrees, randomTress, sizeMemPopulation, hipMemcpyHostToDevice); 

        int sizeDataTest = sizeof(float)*(nrowTest*nvar); /*!< Variable that stores the size in bytes the size of the test data*/
        int sizeDataTestTarget = sizeof(float)*(nrowTest); /*!< Variable that stores the size in bytes the size of the target data */
        
        float *unssenDataTest, *dUnssenDataTest, *unssenDataTestTarget, *hNormalizeData, *dNormalizeData; /*!< This vector pointers to store the individuals of the test data and target data */
        
        unssenDataTest = (float *)malloc(sizeDataTest); /*!< Reserve memory on host*/
        unssenDataTestTarget = (float *)malloc(sizeDataTestTarget); /*!< Reserve memory on host*/
        hNormalizeData= (float *)malloc(sizeMemNormalize);
        
        hipMalloc((void **)&dNormalizeData, sizeMemNormalize);
        hipMalloc((void **)&dUnssenDataTest, sizeDataTest);

        readInpuTestData(path_test, unssenDataTest, nrowTest, nvar);

        hipMemcpy(dUnssenDataTest, unssenDataTest, sizeDataTest, hipMemcpyHostToDevice); 
        
        sizeMemSemanticTest = sizeof(float)*(config.populationSize*nrowTest); /*!< Variable that stores the size in bytes of semantics for the entire population with test data*/

        hipMalloc((void**)&uPushGenes, sizeMemIndividuals);
        hipMalloc((void**)&uStackInd, sizeMemPopulation);  

        float *uSemanticCases, *hSemanticCases, *uSemanticRandomTrees,*hSemanticRandomTrees; /*!< pointer of vectors that contain the semantics of an individual in the population, calculated with the training set and test in generation g and its allocation in GPU*/
        hipMalloc((void**)&uSemanticCases,sizeMemSemanticTest);            
        hipMalloc((void**)&uSemanticRandomTrees,sizeMemSemanticTest);
        hSemanticCases = (float*)malloc(sizeMemSemanticTest);
        hSemanticRandomTrees= (float*)malloc(sizeMemSemanticTest);             

        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, computeSemantics, 0, config.populationSize); /*!< heuristic function used to choose a good block size is to aim at high occupancy*/
        gridSize = (config.populationSize + blockSize - 1) / blockSize; /*!< round up according to array size*/            

        /*!< invokes the GPU to interpret the initial population with data train*/
        computeSemantics<<< gridSize, blockSize >>>(dInitialPopulation, uSemanticCases, individualLength, dUnssenDataTest, nrowTest, nvar, uPushGenes, uStackInd,config.protected_division);
        cudaErrorCheck("computeSemantics");
        hipMemcpy(hSemanticCases,uSemanticCases, sizeMemSemanticTest,hipMemcpyDeviceToHost);

        computeSemantics<<< gridSize, blockSize >>>(dRandomTrees, uSemanticRandomTrees, individualLength, dUnssenDataTest, nrowTest, nvar, uPushGenes, uStackInd,config.protected_division);
        cudaErrorCheck("computeSemantics");
        // hipMemcpy(hSemanticRandomTrees,uSemanticRandomTrees, sizeMemSemanticTest,hipMemcpyDeviceToHost);
        // saveSemantics(logPath,hSemanticRandomTrees,config.populationSize,la,nrowTest,3);
        // hipMemcpy(hSemanticRandomTrees,uSemanticRandomTrees, sizeMemSemanticTest,hipMemcpyDeviceToHost);
        // hipDeviceSynchronize();
        // saveSemantics(logPath,uSemanticRandomTrees,config.populationSize,la,nrowTest,2);
        if(config.normalize==1){
            /*!< invokes the GPU to interpret the random trees with data train*/
            readDataNormalize(logPath,hNormalizeData,config.populationSize,la);
            hipMemcpy(dNormalizeData, hNormalizeData, sizeMemNormalize, hipMemcpyHostToDevice); 
            /*!< invokes the GPU to interpret the random trees with data train*/
            // computeNormalizeData<<< gridSize, blockSize >>>( uSemanticRandomTrees, dNormalizeData, nrowTest);
            cudaErrorCheck("computeNormalizeData");
            printf("en normalize test \n");
            computeNormalizeSemantic<<< gridSize, blockSize >>>(uSemanticRandomTrees, dNormalizeData, nrowTest, config.do_min_max,0);
            cudaErrorCheck("computeNormalizeSemantic");            
        }
        hipMemcpy(hSemanticRandomTrees,uSemanticRandomTrees, sizeMemSemanticTest,hipMemcpyDeviceToHost);
        saveSemantics(logPath,hSemanticRandomTrees,config.populationSize,la,nrowTest,3);
        // hipMemcpy(hNormalizeData,dNormalizeData, sizeMemNormalize,hipMemcpyDeviceToHost);
        /*!< Create file for saved results of best model with the unseen data*/
        std::ofstream OUT(outFile,ios::out);
        evaluate_data(pathTrace, config.numberGenerations, hSemanticCases, hSemanticRandomTrees, OUT, config.logPath, nrowTest, nvar, config.sigmoid, config);
        
        free(unssenDataTest); 
        free(unssenDataTestTarget);
        free(initPopulation);
        free(randomTress);
        hipFree(dInitialPopulation);
        hipFree(dRandomTrees);
        hipFree(uSemanticCases);
        hipFree(uSemanticRandomTrees);
        hipFree(uPushGenes);
        hipFree(uStackInd);
        hipFree(dNormalizeData);
        free(hNormalizeData);
    }
    hipDeviceReset();
    return 0;
}
